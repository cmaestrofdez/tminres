/*
 * CusparseOperator.cpp
 * Created on: Jun 26, 2012
 *  Author: Santiago Akle
 *
 */

#include "CusparseOperator.hpp"
#include "hipsparse.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <time.h>
CusparseOperator::CusparseOperator(hipsparseHandle_t handle_, std::vector<int> row_ptr, std::vector<int> col_ix, std::vector<double> vals): handle(handle_)
{
   hipError_t cudaStat;
   //Set the number of non zeros
   nnz = vals.size(); 
   //Set the number of rows
   n   = row_ptr.size()-1;

   hipsparseStatus_t err = hipsparseCreateMatDescr(&matDesc); 
   if(err!=HIPSPARSE_STATUS_SUCCESS)
   {
      std::cerr << "Unable to allocate matrix descriptor\n";
      throw err;

   }
    
   //Set the type to hermitian and the index base to one 
   hipsparseSetMatType (matDesc, HIPSPARSE_MATRIX_TYPE_GENERAL);  
   hipsparseSetMatIndexBase (matDesc, HIPSPARSE_INDEX_BASE_ONE) ;

   // Allocate the space for the vectors that define the matrix 
   cudaStat = hipMalloc((void**)&csrValA,sizeof(double)*nnz);
   if(cudaStat != hipSuccess)
   {
      std::cerr << "Unable to allocate device memory for operator\n";
      throw cudaStat;
   }
  // Space for the values
   cudaStat = hipMalloc((void**)&csrColIndA,sizeof(int)*nnz);
   if(cudaStat != hipSuccess)
   {
      std::cerr << "Unable to allocate device memory for operator\n";
      throw cudaStat;
   }
   //Space for the pointers to the row starts
   cudaStat = hipMalloc((void**)&csrRowPtrA,sizeof(int)*(n+1));
   if(cudaStat != hipSuccess)
   {
      std::cerr << "Unable to allocate device memory for operator\n";
      throw cudaStat;
   }

  //Load the column indices to the gpu
   cudaStat = hipMemcpy(csrColIndA,&col_ix[0],sizeof(int)*col_ix.size(),hipMemcpyHostToDevice);
   if(cudaStat != hipSuccess)
   {
      std::cerr << "Unable to allocate device memory for operator\n";
      throw cudaStat;
   }
 
   //Load the matrix values to the gpu
   cudaStat = hipMemcpy(csrValA,&vals[0],sizeof(double)*vals.size(),hipMemcpyHostToDevice); 
   if(cudaStat != hipSuccess)
   {
      std::cerr << "Unable to allocate device memory for operator\n";
      throw cudaStat;
   }

   //Load the row start poitners to the gpu
   cudaStat = hipMemcpy(csrRowPtrA,&row_ptr[0],sizeof(int)*row_ptr.size(),hipMemcpyHostToDevice); 
   if(cudaStat != hipSuccess)
   {
      std::cerr << "Unable to allocate device memory for operator\n";
      throw cudaStat;
   }
   
}

CusparseOperator::CusparseOperator(hipsparseHandle_t handle_): handle(handle_)
{}

CusparseOperator::~CusparseOperator()
{
    hipFree(csrValA);
    hipFree(csrColIndA);
    hipFree(csrRowPtrA);
    hipsparseStatus_t cusparseStatus = hipsparseDestroyMatDescr(matDesc);
    if(cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
       std::cerr << "Unable to free descriptor\n";
       throw cusparseStatus;
    }
}


void CusparseOperator::Apply(const CusparseVector & x, CusparseVector & y) const
{
    const double alpha = 1;
    const double beta  = 0;
    hipsparseStatus_t cusparseStatus = hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,n,n,nnz,&alpha,matDesc,csrValA,csrRowPtrA,csrColIndA,x.d_v,&beta,y.d_v);  
    if(cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
       std::cerr << "Unable to execute matrix vector product, Error: "<< cusparseStatus<<"\n";
       throw cusparseStatus;
    }

}
