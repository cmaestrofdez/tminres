/*
 * CusparseSymmetricOperator.cpp
 * Created on: Sep 16, 2012
 *  Author: Santiago Akle
 *
 */

#include "CusparseSymmetricOperator.hpp"
#include "hipsparse.h"
#include <hip/hip_runtime.h>
#include <vector>

CusparseSymmetricOperator::CusparseSymmetricOperator(hipsparseHandle_t handle_, std::vector<int> row_ptr, std::vector<int> col_ix, std::vector<double> vals): CusparseOperator(handle_)
{
   hipError_t cudaStat;
   //Set the number of non zeros
   nnz = vals.size(); 
   //Set the number of rows
   n   = row_ptr.size()-1;

   hipsparseStatus_t err = hipsparseCreateMatDescr(&matDesc); 
   if(err!=HIPSPARSE_STATUS_SUCCESS)
   {
      std::cerr << "Unable to allocate matrix descriptor\n";
      throw err;

   }
    
   //Set the type to hermitian and the index base to one 
   hipsparseSetMatType (matDesc, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);  
   hipsparseSetMatIndexBase (matDesc, HIPSPARSE_INDEX_BASE_ONE) ;
   //Assume that the data corresponds to the upper triangular section of 
   //the matrix.
   hipsparseSetMatFillMode(matDesc, HIPSPARSE_FILL_MODE_UPPER);

   // Allocate the space for the vectors that define the matrix 
   cudaStat = hipMalloc((void**)&csrValA,sizeof(double)*nnz);
   if(cudaStat != hipSuccess)
   {
      std::cerr << "Unable to allocate device memory for operator\n";
      throw cudaStat;
   }
  // Space for the values
   cudaStat = hipMalloc((void**)&csrColIndA,sizeof(int)*nnz);
   if(cudaStat != hipSuccess)
   {
      std::cerr << "Unable to allocate device memory for operator\n";
      throw cudaStat;
   }
   //Space for the pointers to the row starts
   cudaStat = hipMalloc((void**)&csrRowPtrA,sizeof(int)*(n+1));
   if(cudaStat != hipSuccess)
   {
      std::cerr << "Unable to allocate device memory for operator\n";
      throw cudaStat;
   }

  //Load the column indices to the gpu
   cudaStat = hipMemcpy(csrColIndA,&col_ix[0],sizeof(int)*col_ix.size(),hipMemcpyHostToDevice);
   if(cudaStat != hipSuccess)
   {
      std::cerr << "Unable to allocate device memory for operator\n";
      throw cudaStat;
   }
 
   //Load the matrix values to the gpu
   cudaStat = hipMemcpy(csrValA,&vals[0],sizeof(double)*vals.size(),hipMemcpyHostToDevice); 
   if(cudaStat != hipSuccess)
   {
      std::cerr << "Unable to allocate device memory for operator\n";
      throw cudaStat;
   }

   //Load the row start poitners to the gpu
   cudaStat = hipMemcpy(csrRowPtrA,&row_ptr[0],sizeof(int)*row_ptr.size(),hipMemcpyHostToDevice); 
   if(cudaStat != hipSuccess)
   {
      std::cerr << "Unable to allocate device memory for operator\n";
      throw cudaStat;
   }
   
}

